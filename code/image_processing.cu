#include "hip/hip_runtime.h"

#include "image_processing.cuh"
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void sobel_filter(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= 1 && x < width-1 && y >= 1 && y < height-1) {
        int gx = -input[(y-1)*width + (x-1)] - 2*input[y*width + (x-1)] - input[(y+1)*width + (x-1)]
                 + input[(y-1)*width + (x+1)] + 2*input[y*width + (x+1)] + input[(y+1)*width + (x+1)];
        
        int gy = -input[(y-1)*width + (x-1)] - 2*input[(y-1)*width + x] - input[(y-1)*width + (x+1)]
                 + input[(y+1)*width + (x-1)] + 2*input[(y+1)*width + x] + input[(y+1)*width + (x+1)];
        
        output[y*width + x] = min(255, (int)sqrtf(gx*gx + gy*gy));
    }
}